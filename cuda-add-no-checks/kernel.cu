
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addTwoVectors_kernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3], c[4]);

    return 0;
}

// 1. Pass data to GPU
// 2. Tell GPU what to do with data.
// 3. Wait for response
// 4. Profit
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addTwoVectors_kernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
       // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
